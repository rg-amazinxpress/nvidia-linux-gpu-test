#include <hip/hip_runtime.h>
#include <stdio.h>
int main() {
    const size_t MB = 256;
    const int iters = 50;
    for (int i=0; i<iters; i++) {
        void* ptr;
        hipError_t err = hipMalloc(&ptr, MB*1024*1024);
        if (err != hipSuccess) {
            printf("hipMalloc failed at iter %d: %s\n", i, hipGetErrorString(err));
            return 1;
        }
        hipMemset(ptr, 0xA5, MB*1024*1024);
        hipFree(ptr);
    }
    printf("VRAM allocation/free test completed successfully.\n");
    return 0;
}
